
/*******************************
1 - Install nvidia-cuda-toolkit

2 - Compile this program using:

     nvcc add.cu -o add_cuda.out
*******************************/

/*
Program that runs block size dynamically. As the number of threads increase,
the number of blocks is determined as a function of threads and input size.
This provides a constant optimal performance even though the number of threads 
change 
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cstdio>



//CUDA kernel to add elements of the matrix
// __global__ converts a function into a CUDA kernel
__global__
void add(int n, float *x, float *y)
{
  // index of the current thread within the block
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // number of threads in a block
  int stride = blockDim.x * gridDim.x;

  // run each addition on a separate thread
  for (int i = index; i < n; i+=stride)
      y[i] = x[i] + y[i];
}




int main(void)
{
  for(int t = 32; t <= 1024; t+=32)
  {
    int N = 1<<24; // 2^24 elements

    // Memory allocation in CUDA is done with cudaMallocManaged( , )
    float *x; float *y;
    hipMallocManaged( &x, N*sizeof(float) );
    hipMallocManaged( &y, N*sizeof(float) );


    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }


    std::clock_t start = clock();
    // Launch the 'add' kernel, which invokes it in the GPU
    int blockSize = t;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::cout << "BlockSize = " << t << ",NumBlocks = " << numBlocks << "\n";
    add<<<numBlocks,blockSize>>>(N, x, y);
    

    // Wait for the GPU to synchronize before accessign through host(CPU)
    hipDeviceSynchronize();

    std::clock_t stop = clock();
    int duration = 1000 * (stop - start) / (double)CLOCKS_PER_SEC;
    //std::cout << "Running time using " << t << " threads = " << duration << "\n"; 
    std::cout << duration << "\n";

    // Check for errors (all values should be 3.0f)
    /*float maxError = 0.0f;
    for (int i = 0; i < N; i++)
      maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;
    */

    // Deallocating memory using cudaFree()
    hipFree(x);
    hipFree(y);
  }

  return 0;
}

